#include <cupti_target.h>
#include <cupti_profiler_target.h>
#include <nvperf_host.h>
#include <hip/hip_runtime.h>
#include <string>
#include <stdio.h>
#include <stdlib.h>
#include <Metric.h>
#include <Eval.h>
#include <FileOp.h>

#define L2_CACHE_READS  "lts__t_sectors_op_read.sum"
#define L2_CACHE_WRITES "lts__t_sectors_op_write.sum"

#define EXIT_WAIVED 2

#define NVPW_API_CALL(apiFuncCall)                                             \
do {                                                                           \
    NVPA_Status _status = apiFuncCall;                                         \
    if (_status != NVPA_STATUS_SUCCESS) {                                      \
        fprintf(stderr, "%s:%d: error: function %s failed with error %d.\n",   \
                __FILE__, __LINE__, #apiFuncCall, _status);                    \
        exit(EXIT_FAILURE);                                                    \
    }                                                                          \
} while (0)

#define CUPTI_API_CALL(apiFuncCall)                                            \
do {                                                                           \
    CUptiResult _status = apiFuncCall;                                         \
    if (_status != CUPTI_SUCCESS) {                                            \
        const char *errstr;                                                    \
        cuptiGetResultString(_status, &errstr);                                \
        fprintf(stderr, "%s:%d: error: function %s failed with error %s.\n",   \
                __FILE__, __LINE__, #apiFuncCall, errstr);                     \
        exit(EXIT_FAILURE);                                                    \
    }                                                                          \
} while (0)

#define DRIVER_API_CALL(apiFuncCall)                                           \
do {                                                                           \
    hipError_t _status = apiFuncCall;                                            \
    if (_status != hipSuccess) {                                             \
        fprintf(stderr, "%s:%d: error: function %s failed with error %d.\n",   \
                __FILE__, __LINE__, #apiFuncCall, _status);                    \
        exit(EXIT_FAILURE);                                                    \
    }                                                                          \
} while (0)

#define RUNTIME_API_CALL(apiFuncCall)                                          \
do {                                                                           \
    hipError_t _status = apiFuncCall;                                         \
    if (_status != hipSuccess) {                                              \
        fprintf(stderr, "%s:%d: error: function %s failed with error %s.\n",   \
                __FILE__, __LINE__, #apiFuncCall, hipGetErrorString(_status));\
        exit(EXIT_FAILURE);                                                    \
    }                                                                          \
} while (0)

#define HANDLE_COMPATABILITY(params) \
do { \
    if (params.isSupported != CUPTI_PROFILER_CONFIGURATION_SUPPORTED) { \
        std::cerr << "Unable to profile on device " << deviceNum << ::std::endl; \
        if (params.architecture == CUPTI_PROFILER_CONFIGURATION_UNSUPPORTED) { \
            std::cerr << "\tdevice architecture is not supported" << ::std::endl; \
        } \
        if (params.sli == CUPTI_PROFILER_CONFIGURATION_UNSUPPORTED) { \
            std::cerr << "\tdevice sli configuration is not supported" << ::std::endl; \
        } \
        if (params.vGpu == CUPTI_PROFILER_CONFIGURATION_UNSUPPORTED) { \
            std::cerr << "\tdevice vgpu configuration is not supported" << ::std::endl; \
        } \
        else if (params.vGpu == CUPTI_PROFILER_CONFIGURATION_DISABLED) { \
            std::cerr << "\tdevice vgpu configuration disabled profiling support" << ::std::endl; \
        } \
        if (params.confidentialCompute == CUPTI_PROFILER_CONFIGURATION_UNSUPPORTED) { \
            std::cerr << "\tdevice confidential compute configuration is not supported" << ::std::endl; \
        } \
        if (params.cmp == CUPTI_PROFILER_CONFIGURATION_UNSUPPORTED) { \
            std::cerr << "\tNVIDIA Crypto Mining Processors (CMP) are not supported" << ::std::endl; \
        } \
        exit(EXIT_WAIVED); \
    } \
} while (0)

bool run(std::vector<uint8_t>&configImage, std::vector<uint8_t>&counterDataScratchBuffer, std::vector<uint8_t>&counterDataImage, CUpti_ProfilerReplayMode profilerReplayMode, CUpti_ProfilerRange profilerRange) {
    hipCtx_t cuContext;
    DRIVER_API_CALL(hipCtxGetCurrent(&cuContext));
    CUpti_Profiler_BeginSession_Params beginSessionParams = {CUpti_Profiler_BeginSession_Params_STRUCT_SIZE};
    CUpti_Profiler_SetConfig_Params setConfigParams = {CUpti_Profiler_SetConfig_Params_STRUCT_SIZE};
    CUpti_Profiler_EnableProfiling_Params enableProfilingParams = {CUpti_Profiler_EnableProfiling_Params_STRUCT_SIZE};
    CUpti_Profiler_DisableProfiling_Params disableProfilingParams = {CUpti_Profiler_DisableProfiling_Params_STRUCT_SIZE};
    CUpti_Profiler_PushRange_Params pushRangeParams = {CUpti_Profiler_PushRange_Params_STRUCT_SIZE};
    CUpti_Profiler_PopRange_Params popRangeParams = {CUpti_Profiler_PopRange_Params_STRUCT_SIZE};

    beginSessionParams.ctx = NULL;
    beginSessionParams.counterDataImageSize = counterDataImage.size();
    beginSessionParams.pCounterDataImage = &counterDataImage[0];
    beginSessionParams.counterDataScratchBufferSize = counterDataScratchBuffer.size();
    beginSessionParams.pCounterDataScratchBuffer = &counterDataScratchBuffer[0];
    beginSessionParams.range = profilerRange;
    beginSessionParams.replayMode = profilerReplayMode;
    beginSessionParams.maxRangesPerPass = 1;
    beginSessionParams.maxLaunchesPerPass = 1;

    CUPTI_API_CALL(cuptiProfilerBeginSession(&beginSessionParams));

    setConfigParams.pConfig = &configImage[0];
    setConfigParams.configSize = configImage.size();

    setConfigParams.passIndex = 0;
    setConfigParams.minNestingLevel = 1;
    setConfigParams.numNestingLevels = 1;
    CUPTI_API_CALL(cuptiProfilerSetConfig(&setConfigParams));
    /* User takes the resposiblity of replaying the kernel launches */
    CUpti_Profiler_BeginPass_Params beginPassParams = {CUpti_Profiler_BeginPass_Params_STRUCT_SIZE};
    CUpti_Profiler_EndPass_Params endPassParams = {CUpti_Profiler_EndPass_Params_STRUCT_SIZE};


    // code will run each replay here
    do {
        CUPTI_API_CALL(cuptiProfilerBeginPass(&beginPassParams));

        {
            CUPTI_API_CALL(cuptiProfilerEnableProfiling(&enableProfilingParams));
            std::string rangeName = "userrangeA";
            pushRangeParams.pRangeName = rangeName.c_str();
            CUPTI_API_CALL(cuptiProfilerPushRange(&pushRangeParams));

            {
                // Actual function to run is here
                system("echo hello world");
            }

            CUPTI_API_CALL(cuptiProfilerPopRange(&popRangeParams));
            CUPTI_API_CALL(cuptiProfilerDisableProfiling(&disableProfilingParams));
        }

        CUPTI_API_CALL(cuptiProfilerEndPass(&endPassParams));
    } while (!endPassParams.allPassesSubmitted);


    CUpti_Profiler_FlushCounterData_Params flushCounterDataParams = {CUpti_Profiler_FlushCounterData_Params_STRUCT_SIZE};
    CUPTI_API_CALL(cuptiProfilerFlushCounterData(&flushCounterDataParams));
    CUpti_Profiler_UnsetConfig_Params unsetConfigParams = {CUpti_Profiler_UnsetConfig_Params_STRUCT_SIZE};
    CUPTI_API_CALL(cuptiProfilerUnsetConfig(&unsetConfigParams));
    CUpti_Profiler_EndSession_Params endSessionParams = {CUpti_Profiler_EndSession_Params_STRUCT_SIZE};
    CUPTI_API_CALL(cuptiProfilerEndSession(&endSessionParams));

    return true;
}

bool CreateCounterDataImage(
    std::vector<uint8_t>& counterDataImage,
    std::vector<uint8_t>& counterDataScratchBuffer,
    std::vector<uint8_t>& counterDataImagePrefix)
{
    CUpti_Profiler_CounterDataImageOptions counterDataImageOptions;
    counterDataImageOptions.pCounterDataPrefix = &counterDataImagePrefix[0];
    counterDataImageOptions.counterDataPrefixSize = counterDataImagePrefix.size();
    counterDataImageOptions.maxNumRanges = 1;
    counterDataImageOptions.maxNumRangeTreeNodes = 1;
    counterDataImageOptions.maxRangeNameLength = 64;

    CUpti_Profiler_CounterDataImage_CalculateSize_Params calculateSizeParams = {CUpti_Profiler_CounterDataImage_CalculateSize_Params_STRUCT_SIZE};

    calculateSizeParams.pOptions = &counterDataImageOptions;
    calculateSizeParams.sizeofCounterDataImageOptions = CUpti_Profiler_CounterDataImageOptions_STRUCT_SIZE;

    CUPTI_API_CALL(cuptiProfilerCounterDataImageCalculateSize(&calculateSizeParams));

    CUpti_Profiler_CounterDataImage_Initialize_Params initializeParams = {CUpti_Profiler_CounterDataImage_Initialize_Params_STRUCT_SIZE};
    initializeParams.sizeofCounterDataImageOptions = CUpti_Profiler_CounterDataImageOptions_STRUCT_SIZE;
    initializeParams.pOptions = &counterDataImageOptions;
    initializeParams.counterDataImageSize = calculateSizeParams.counterDataImageSize;

    counterDataImage.resize(calculateSizeParams.counterDataImageSize);
    initializeParams.pCounterDataImage = &counterDataImage[0];
    CUPTI_API_CALL(cuptiProfilerCounterDataImageInitialize(&initializeParams));

    CUpti_Profiler_CounterDataImage_CalculateScratchBufferSize_Params scratchBufferSizeParams = {CUpti_Profiler_CounterDataImage_CalculateScratchBufferSize_Params_STRUCT_SIZE};
    scratchBufferSizeParams.counterDataImageSize = calculateSizeParams.counterDataImageSize;
    scratchBufferSizeParams.pCounterDataImage = initializeParams.pCounterDataImage;
    CUPTI_API_CALL(cuptiProfilerCounterDataImageCalculateScratchBufferSize(&scratchBufferSizeParams));

    counterDataScratchBuffer.resize(scratchBufferSizeParams.counterDataScratchBufferSize);

    CUpti_Profiler_CounterDataImage_InitializeScratchBuffer_Params initScratchBufferParams = {CUpti_Profiler_CounterDataImage_InitializeScratchBuffer_Params_STRUCT_SIZE};
    initScratchBufferParams.counterDataImageSize = calculateSizeParams.counterDataImageSize;

    initScratchBufferParams.pCounterDataImage = initializeParams.pCounterDataImage;
    initScratchBufferParams.counterDataScratchBufferSize = scratchBufferSizeParams.counterDataScratchBufferSize;
    initScratchBufferParams.pCounterDataScratchBuffer = &counterDataScratchBuffer[0];

    CUPTI_API_CALL(cuptiProfilerCounterDataImageInitializeScratchBuffer(&initScratchBufferParams));

    return true;
}

int main(int argc, char* argv[]) {

    hipDevice_t cuDevice;
    std::vector<std::string> metrics;
    std::vector<uint8_t> counterDataImagePrefix;
    std::vector<uint8_t> configImage;
    std::vector<uint8_t> counterDataImage;
    std::vector<uint8_t> counterDataScratchBuffer;
    std::vector<uint8_t> counterAvailabilityImage;

    std::string CounterDataFileName("data.counterdata");
    std::string CounterDataSBFileName("data.counterdataSB");

    CUpti_ProfilerReplayMode profilerReplayMode = CUPTI_UserReplay;
    CUpti_ProfilerRange profilerRange = CUPTI_UserRange;

    char *metricName;

    int deviceNum = 0;
    if (argc > 1) {
        deviceNum = atoi(argv[1]);
    }

    DRIVER_API_CALL(hipInit(0));
    DRIVER_API_CALL(hipDeviceGet(&cuDevice, deviceNum));

    CUpti_Profiler_Initialize_Params profilerInitializeParams = {CUpti_Profiler_Initialize_Params_STRUCT_SIZE};
    CUPTI_API_CALL(cuptiProfilerInitialize(&profilerInitializeParams));

    CUpti_Profiler_DeviceSupported_Params params = {CUpti_Profiler_DeviceSupported_Params_STRUCT_SIZE};
    params.cuDevice = deviceNum;
    CUPTI_API_CALL(cuptiProfilerDeviceSupported(&params));

    // run the check to make sure we can run what we need to
    HANDLE_COMPATABILITY(params);

    // push the requested metrics, if not available use the defaults
    if (argc > 2) {
        metricName = strtok(argv[2], ",");
        while(metricName != NULL) {
            metrics.push_back(metricName);
            metricName = strtok(NULL, ",");
        }
    } else {
        metrics.push_back(L2_CACHE_READS);
        metrics.push_back(L2_CACHE_WRITES);
    }

    std::cout << "---- Metrics ----" << std::endl;
    for (auto &metric : metrics) {
        std::cout << metric << std::endl;
    }

    CUpti_Device_GetChipName_Params getChipNameParams = { CUpti_Device_GetChipName_Params_STRUCT_SIZE };
    getChipNameParams.deviceIndex = deviceNum;
    CUPTI_API_CALL(cuptiDeviceGetChipName(&getChipNameParams));
    std::string chipName(getChipNameParams.pChipName);

    hipCtx_t cuContext;
    DRIVER_API_CALL(hipCtxCreate(&cuContext, 0, cuDevice));

    CUpti_Profiler_GetCounterAvailability_Params getCounterAvailabilityParams = {CUpti_Profiler_GetCounterAvailability_Params_STRUCT_SIZE};
    getCounterAvailabilityParams.ctx = cuContext;
    CUPTI_API_CALL(cuptiProfilerGetCounterAvailability(&getCounterAvailabilityParams));

    counterAvailabilityImage.clear();
    counterAvailabilityImage.resize(getCounterAvailabilityParams.counterAvailabilityImageSize);
    getCounterAvailabilityParams.pCounterAvailabilityImage = counterAvailabilityImage.data();
    CUPTI_API_CALL(cuptiProfilerGetCounterAvailability(&getCounterAvailabilityParams));

    NVPW_InitializeHost_Params initializeHostParams = { NVPW_InitializeHost_Params_STRUCT_SIZE };
    NVPW_API_CALL(NVPW_InitializeHost(&initializeHostParams));

    if(!NV::Metric::Config::GetConfigImage(chipName, metrics, configImage, counterAvailabilityImage.data())) {
        std::cout << "Failed to create configImage" << std::endl;
        exit(EXIT_FAILURE);
    }

    if(!NV::Metric::Config::GetCounterDataPrefixImage(chipName, metrics, counterDataImagePrefix)) {
        std::cout << "Failed to create counterDataImagePrefix" << std::endl;
        exit(EXIT_FAILURE);
    }

    if(!CreateCounterDataImage(counterDataImage, counterDataScratchBuffer, counterDataImagePrefix)) {
        std::cout << "Failed to create counterDataImage" << std::endl;
        exit(EXIT_FAILURE);
    }

    if(!run(configImage, counterDataScratchBuffer, counterDataImage, profilerReplayMode, profilerRange)) {
        std::cout << "Failed to run sample" << std::endl;
        exit(EXIT_FAILURE);
    }

    CUpti_Profiler_DeInitialize_Params profilerDeInitializeParams = {CUpti_Profiler_DeInitialize_Params_STRUCT_SIZE};
    CUPTI_API_CALL(cuptiProfilerDeInitialize(&profilerDeInitializeParams));

    /* Dump counterDataImage in file */
    WriteBinaryFile(CounterDataFileName.c_str(), counterDataImage);
    WriteBinaryFile(CounterDataSBFileName.c_str(), counterDataScratchBuffer);

    /* Evaluation of metrics collected in counterDataImage, this can also be done offline*/
    NV::Metric::Eval::PrintMetricValues("A100", counterDataImage, metrics);

    exit(EXIT_SUCCESS);
}